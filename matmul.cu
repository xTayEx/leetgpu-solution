#include "hip/hip_runtime.h"
#include "helper.cuh"
#include <cassert>
#include <cstdio>
#include <hip/hip_runtime.h>

constexpr int TILE_SIZE = 128;
constexpr int EXPAND_FACTOR = 4;

__global__ void matrix_multiplication_kernel(const float *A, const float *B,
                                             float *C, int M, int N, int K) {
  __shared__ float As[TILE_SIZE][TILE_SIZE / EXPAND_FACTOR]; // (16, 4)
  __shared__ float Bs[TILE_SIZE / EXPAND_FACTOR][TILE_SIZE]; // (4, 16)

  size_t block_row = blockIdx.y * TILE_SIZE;
  size_t block_col = blockIdx.x * TILE_SIZE;

  size_t thread_row = threadIdx.y * EXPAND_FACTOR;
  size_t thread_col = threadIdx.x * EXPAND_FACTOR;

  float sums[EXPAND_FACTOR][EXPAND_FACTOR] = {0.0f};

  // split along the common dimension
  for (int tile_start = 0; tile_start < N; tile_start += TILE_SIZE / EXPAND_FACTOR) {
#pragma unroll
    for (int i = 0; i < EXPAND_FACTOR; ++i) {
      int load_row = block_row + thread_row + i;
      int load_col = tile_start + threadIdx.x;
      if (load_row < M && load_col < N) {
        As[thread_row + i][threadIdx.x] = A[load_row * N + load_col];
        // printf("As[%d][%d]: %f\n", int(thread_row + i), threadIdx.x,
        //        As[thread_row + i][threadIdx.x]);
      } else {
        // printf("FUCK! else!\n");
        As[thread_row + i][threadIdx.x] = 0.0f;
      }
    }

#pragma unroll
    for (int j = 0; j < EXPAND_FACTOR; ++j) {
      int load_row = tile_start + threadIdx.y;
      int load_col = block_col + thread_col + j;
      if (load_row < N && load_col < K) {
        Bs[threadIdx.y][thread_col + j] = B[load_row * K + load_col];
        // printf("Bs[%d][%d]: %f\n", threadIdx.y, int(thread_col + j),
        //        Bs[threadIdx.y][thread_col + j]);
      } else {
        // printf("FUCK! else!\n");
        Bs[threadIdx.y][thread_col + j] = 0.0f;
      }
    }
    __syncthreads();

#pragma unroll
    for (int k = 0; k < (TILE_SIZE / EXPAND_FACTOR); ++k) {
      float a_frag[EXPAND_FACTOR];
      for (int i = 0; i < EXPAND_FACTOR; ++i) {
        a_frag[i] = As[thread_row + i][k];
      }

      float b_frag[EXPAND_FACTOR];
#pragma unroll
      for (int j = 0; j < EXPAND_FACTOR; ++j) {
        b_frag[j] = Bs[k][thread_col + j];
      }

#pragma unroll
      for (int i = 0; i < EXPAND_FACTOR; ++i) {
#pragma unroll
        for (int j = 0; j < EXPAND_FACTOR; ++j) {
          // if (a_frag[i] == 0.0f || b_frag[j] == 0.0f) {
          //   printf("FUCK! a_frag[%d] = %f, b_frag[%d] = %f\n", i, a_frag[i], j, b_frag[j]);
          // }
          sums[i][j] += a_frag[i] * b_frag[j];
          // printf("a_frag[%d] = %f, b_frag[%d] = %f\n", i, a_frag[i], j,
          //        b_frag[j]);
        }
      }
    }

    // 在所有线程都算好之前，任何线程都不准偷跑，不然如果某个线程提前进入下一轮循环，
    // 会覆盖shm上的数据
    __syncthreads();
    // printf("sums[0][0] = %f\n", sums[0][0]);
  }
#pragma unroll
  for (int i = 0; i < EXPAND_FACTOR; ++i) {
#pragma unroll
    for (int j = 0; j < EXPAND_FACTOR; ++j) {
      int global_row = block_row + thread_row + i;
      int global_col = block_col + thread_col + j;
      if (global_row < M && global_col < K) {
        C[global_row * K + global_col] = sums[i][j];
      }
    }
  }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float *A, const float *B, float *C, int M, int N, int K) {
  static_assert(TILE_SIZE >= EXPAND_FACTOR,
                "TILE_SIZE must be greater than or equal to EXPAND_FACTOR");
  static_assert(TILE_SIZE % EXPAND_FACTOR == 0,
                "TILE_SIZE must be divisible by EXPAND_FACTOR");
  dim3 threadsPerBlock(TILE_SIZE / EXPAND_FACTOR, TILE_SIZE / EXPAND_FACTOR);
  dim3 blocksPerGrid((K + TILE_SIZE - 1) / TILE_SIZE,
                     (M + TILE_SIZE - 1) / TILE_SIZE);

  hipMemset(C, 0, sizeof(float) * M * K);
  matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M,
                                                                   N, K);
  hipDeviceSynchronize();
}

int main() {
  // test the solve function
  const int M = 1, N = 1, K = 1;
  float A[M * N] = {0.0f};
  float B[N * K] = {0.0f};
  for (int i = 0; i < M * N; ++i) {
    A[i] = 2.f;
  }
  for (int j = 0; j < N * K; ++j) {
    B[j] = 3.f;
  }
  float C[M * K] = {0};
  float *d_A, *d_B, *d_C;
  CHECK_CUDA(hipMalloc((void **)&d_A, M * N * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_B, N * K * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_C, M * K * sizeof(float)));
  CHECK_CUDA(hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice));
  solve(d_A, d_B, d_C, M, N, K);
  hipMemcpy(C, d_C, M * K * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < K; ++j) {
      printf("%f ", C[i * K + j]);
    }
    printf("\n");
  }
}
