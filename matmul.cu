#include "hip/hip_runtime.h"
#include "helper.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define TILE_SIZE 16

__global__ void matrix_multiplication_kernel(const float *A, const float *B,
                                             float *C, int M, int N, int K) {
  __shared__ float As[TILE_SIZE][TILE_SIZE];
  __shared__ float Bs[TILE_SIZE][TILE_SIZE];

  size_t row = blockIdx.y * TILE_SIZE;
  size_t col = blockIdx.x * TILE_SIZE;

  size_t ty = threadIdx.y;
  size_t tx = threadIdx.x;

  int m = row + ty;
  int k = col + tx;

  float sum = 0.0f;

  for (int n_start = 0; n_start < N; n_start += TILE_SIZE) {
    int a_col = n_start + tx;
    if (m < M && a_col < N) {
      As[ty][tx] = A[m * N + a_col];
    } else {
      As[ty][tx] = 0.0f;
    }

    int b_row = n_start + ty;
    if (k < K && b_row < N) {
      Bs[ty][tx] = B[b_row * K + k];
    } else {
      Bs[ty][tx] = 0.0f;
    }

    __syncthreads();

    for (int i = 0; i < TILE_SIZE; ++i) {
      sum += As[ty][i] * Bs[i][tx];
    }

    __syncthreads();
  }

  if (m < M && k < K) {
    C[m * K + k] = sum;
  }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float *A, const float *B, float *C, int M, int N, int K) {
  dim3 threadsPerBlock(16, 16);
  dim3 blocksPerGrid((K + TILE_SIZE - 1) / TILE_SIZE,
                     (M + TILE_SIZE - 1) / TILE_SIZE);

  hipMemset(C, 0, sizeof(float) * M * K);
  matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M,
                                                                   N, K);
  hipDeviceSynchronize();
}


int main() {
  // test the solve function
  const int M = 2, N = 2, K = 2;
  float A[M * N] = {1, 2, 3, 4};
  float B[M * N] = {5, 6, 7, 8};
  float C[M * K] = {0};
  float *d_A, *d_B, *d_C;
  CHECK_CUDA(hipMalloc((void **)&d_A, M * N * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_B, N * K * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_C, M * K * sizeof(float)));
  CHECK_CUDA(hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice));
  solve(d_A, d_B, d_C, M, N, K);
  hipMemcpy(C, d_C, M * K * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < K; ++j) {
      printf("%f ", C[i * K + j]);
    }
    printf("\n");
  }
}
