#include "hip/hip_runtime.h"
#include "helper.cuh"
#include <cstdio>
#include <hip/hip_runtime.h>

constexpr int TILE_SIZE = 16;
constexpr int EXPAND_FACTOR = 4;

__global__ void matrix_multiplication_kernel(const float *A, const float *B,
                                             float *C, int M, int N, int K) {
  __shared__ float As[TILE_SIZE * EXPAND_FACTOR][TILE_SIZE];
  __shared__ float Bs[TILE_SIZE][TILE_SIZE * EXPAND_FACTOR];

  size_t block_row = blockIdx.y * TILE_SIZE;
  size_t block_col = blockIdx.x * TILE_SIZE;

  size_t thread_row = block_row * EXPAND_FACTOR;
  size_t thread_col = block_col * EXPAND_FACTOR;

  float sums[EXPAND_FACTOR][EXPAND_FACTOR] = {0.0f};

  // split along the common dimension
  for (int tile_start = 0; tile_start < N; tile_start += TILE_SIZE) {
#pragma unroll
    for (int i = 0; i < EXPAND_FACTOR; ++i) {
      int load_row = block_row + thread_row + i;
      int load_col = tile_start + threadIdx.x;
      if (load_row < M && load_col < N) {
        As[thread_row + i][threadIdx.x] = A[load_row * N + load_col];
      } else {
        As[thread_row + i][threadIdx.x] = 0.0f;
      }
    }

#pragma unroll
    for (int j = 0; j < EXPAND_FACTOR; ++j) {
      int load_row = tile_start + threadIdx.y;
      int load_col = block_col + thread_col + j;
      if (load_row < N && load_col < K) {
        Bs[threadIdx.y][thread_col + j] = B[load_row * K + load_col];
      } else {
        Bs[threadIdx.y][thread_col + j] = 0.0f;
      }
    }
    __syncthreads();

#pragma unroll
    for (int k = 0; k < TILE_SIZE; ++k) {
      float a_frag[EXPAND_FACTOR];
      for (int i = 0; i < EXPAND_FACTOR; ++i) {
        a_frag[i] = As[thread_row + i][k];
      }

      float b_frag[EXPAND_FACTOR];
#pragma unroll
      for (int j = 0; j < EXPAND_FACTOR; ++j) {
        b_frag[j] = Bs[k][thread_col + j];
      }

#pragma unroll
      for (int i = 0; i < EXPAND_FACTOR; ++i) {
#pragma unroll
        for (int j = 0; j < EXPAND_FACTOR; ++j) {
          sums[i][j] += a_frag[i] * b_frag[j];
        }
      }
    }

    __syncthreads();

#pragma unroll
    for (int i = 0; i < EXPAND_FACTOR; ++i) {
#pragma unroll
      for (int j = 0; j < EXPAND_FACTOR; ++j) {
        int global_row = block_row + thread_row + i;
        int global_col = block_col + thread_col + j;
        if (global_row < M && global_col < K) {
          C[global_row * K + global_col] = sums[i][j];
        }
      }
    }
  }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
void solve(const float *A, const float *B, float *C, int M, int N, int K) {
  dim3 threadsPerBlock(TILE_SIZE / EXPAND_FACTOR, TILE_SIZE / EXPAND_FACTOR);
  dim3 blocksPerGrid((K + TILE_SIZE - 1) / TILE_SIZE,
                     (M + TILE_SIZE - 1) / TILE_SIZE);

  hipMemset(C, 0, sizeof(float) * M * K);
  matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M,
                                                                   N, K);
  hipDeviceSynchronize();
}

int main() {
  // test the solve function
  const int M = 32, N = 32, K = 32;
  float A[M * N] = {0.0f};
  float B[N * K] = {0.0f};
  for (int i = 0; i < M * N; ++i) {
    A[i] = i + 1;
  }
  for (int j = 0; j < N * K; ++j) {
    B[j] = j + 1;
  }
  float C[M * K] = {0};
  float *d_A, *d_B, *d_C;
  CHECK_CUDA(hipMalloc((void **)&d_A, M * N * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_B, N * K * sizeof(float)));
  CHECK_CUDA(hipMalloc((void **)&d_C, M * K * sizeof(float)));
  CHECK_CUDA(hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_B, B, N * K * sizeof(float), hipMemcpyHostToDevice));
  solve(d_A, d_B, d_C, M, N, K);
  hipMemcpy(C, d_C, M * K * sizeof(float), hipMemcpyDeviceToHost);
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < K; ++j) {
      printf("%f ", C[i * K + j]);
    }
    printf("\n");
  }
}
